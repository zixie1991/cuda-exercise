#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

int main() {
  hipError_t cuda_status;
  int num;
  cuda_status = hipGetDeviceCount(&num);
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipGetDeviceCount error\n");
    return -1;
  }

  hipDeviceProp_t prop;
  for (int i = 0; i < num; i++) {
    cuda_status = hipGetDeviceProperties(&prop, i);
    if (cuda_status != hipSuccess) {
      fprintf(stderr, "hipGetDeviceProperties error\n");
      continue;
    }

    printf("%d\n", i);
    printf("Device name: %s\n", prop.name);
    printf("Device memory: %ld\n", prop.totalGlobalMem);
    printf("Version: %d.%d\n", prop.major, prop.minor);
    printf("clockRate: %d\n", prop.clockRate);
    printf("multiProcessorCount: %d\n", prop.multiProcessorCount);
  }

  return 0;
}
